#include <stdio.h>
#include <hip/hip_runtime.h>
 

__global__ void square_cuda(float *a, int N) {
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  if (idx<N) a[idx] = a[idx] * a[idx];
}

extern "C" {

void square(float *a, int N) {
  float* a_d;
  size_t size = N * sizeof(float);
  hipMalloc((void **) &a_d, size);   
  hipMemcpy(a_d, a, size, hipMemcpyHostToDevice);

  int block_size = 4;
  int n_blocks = N/block_size + (N%block_size == 0 ? 0:1);
  square_cuda <<< n_blocks, block_size >>> (a_d, N);

  hipMemcpy(a, a_d, size, hipMemcpyDeviceToHost);
  hipFree(a_d);
}

}
